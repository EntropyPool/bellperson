#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "gpu_support.h"
#include "fields/multiexp.cuh"
#include "fields/fft.cuh"
#include "interface.hpp"

template<typename T>
static uint64_t multiexp_chunk_size(InputParameters<T> p) {
    size_t free, total;
    CUDA_CHECK(hipMemGetInfo(&free, &total));

    size_t bucket_len = 1 << p.window_size;
    size_t buckets_size = sizeof(projective<T>) * (2 * p.core_count * bucket_len);
    size_t results_size = sizeof(projective<T>) * (2 * p.core_count);

    size_t usable = free - buckets_size - results_size - 512 * 1024 * 1024;
    if (usable < 0) {
        return 0;
    }

    return usable / (sizeof(affine<T>) + sizeof(Fr));
}

template<typename T>
static State multiexp_cuda(InputParameters<T> p) {
    // initialize cuda
    hipSetDevice(p.cuda_info.device_id);
    hipCtx_t context;
    hipCtxCreate(&context, 0, p.cuda_info.device_id);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    affine<T> *d_bases;
    projective<T> *d_buckets;
    projective<T> *d_results;
    Fr *d_exps;

    size_t bases_size = sizeof(affine<T>) * p.n;
    size_t bucket_len = 1 << p.window_size;
    size_t buckets_size = sizeof(projective<T>) * (2 * p.core_count * bucket_len);
    size_t results_size = sizeof(projective<T>) * (2 * p.core_count);
    size_t exps_size = sizeof(Fr) * p.n;

    {
        printf("n: %d, num_groups: %d, num_windows: %d, window_size: %d\n",
                p.n, p.num_groups, p.num_windows, p.window_size);
        printf("core_count = %d\n", p.core_count);
        printf("GPU memory size: %llu\n", bases_size + buckets_size + results_size + exps_size);
    }

    CUDA_CHECK(hipMalloc((void**)&d_bases, bases_size));
    CUDA_CHECK(hipMalloc((void**)&d_buckets, buckets_size));
    CUDA_CHECK(hipMalloc((void**)&d_results, results_size));
    CUDA_CHECK(hipMalloc((void**)&d_exps, exps_size));

    CUDA_CHECK(hipMemcpyAsync(d_bases, p.bases, bases_size, hipMemcpyHostToDevice));    
    CUDA_CHECK(hipMemcpyAsync(d_exps, p.exps, exps_size, hipMemcpyHostToDevice));    
    instantiate_constants();
    CUDA_CHECK(hipDeviceSynchronize());

    hipEventRecord(start, 0);

    //size_t heap = sizeof(projective<T>) * ((1 << p.window_size) - 1) + 4096;
    //CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heap));

    // Call cuda
    constexpr uint32_t threadsPerBlock = 256; 
    //uint32_t blocksPerGrid = (2 * p.core_count + threadsPerBlock - 1) / threadsPerBlock;
    uint32_t blocksPerGrid = (uint32_t)ceil((2 * p.core_count) / threadsPerBlock);
    bellman_multiexp<T><<<blocksPerGrid, threadsPerBlock>>>
                (d_bases,
                 d_buckets,
                 d_results,
                 d_exps,
                 p.n,
                 p.num_groups,
                 p.num_windows,
                 p.window_size);

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed=0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Multiexp Kernel time: %fms\n", elapsed);


    CUDA_CHECK(hipMemcpyAsync(p.results, d_results, results_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_bases));
    CUDA_CHECK(hipFree(d_buckets));
    CUDA_CHECK(hipFree(d_exps));


    CUDA_CHECK(hipFree(d_results));

    hipCtxDestroy(context);
    return Compute_Ok;
} 

State radix_fft_cuda(FFTInputParameters p) {
    // initialize cuda
    hipSetDevice(p.cuda_info.device_id);
    hipCtx_t context;
    hipCtxCreate(&context, 0, p.cuda_info.device_id);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate 
    Fr *d_x;
    Fr *d_y;
    Fr *d_pq;
    Fr *d_omegas;

    size_t x_size = sizeof(Fr) * p.n;
    size_t y_size = sizeof(Fr) * p.n;
    size_t pq_size = sizeof(Fr) * (1 << p.max_deg >> 1);
    size_t omegas_size = sizeof(Fr) * 32;

    {
        //printf("CUDA parameters:\n");
        //printf("pq_size: %d, Fr size: %d, omegas_size: %d\n", pq_size, sizeof(Fr), omegas_size);
        //printf("threads perblock: %d, blocks per grid: %d\n", threadsPerBlock, blocksPerGrid);
        //printf("x/y_size: %d, n: %d\n", x_size, p.n);
        //printf("deg: %d, max_deg %d, lgp: %d, u_size %d\n", p.deg, p.max_deg, p.lgp, u_size);
    }

    CUDA_CHECK(hipMalloc((void**)&d_x, x_size));
    CUDA_CHECK(hipMalloc((void**)&d_y, y_size));
    CUDA_CHECK(hipMalloc((void**)&d_pq, pq_size));
    CUDA_CHECK(hipMalloc((void**)&d_omegas, omegas_size));

    CUDA_CHECK(hipMemcpyAsync(d_x, p.x, x_size, hipMemcpyHostToDevice));    
    CUDA_CHECK(hipMemcpyAsync(d_pq, p.pq, pq_size, hipMemcpyHostToDevice));    
    CUDA_CHECK(hipMemcpyAsync(d_omegas, p.omegas, omegas_size, hipMemcpyHostToDevice));    


    {
        //printf("CUDA FFT input parameters:\n");
        //printf("N: %ld\n", p.n);
        //printf("x/y size: %ld\n", x_size);
        //printf("pq size: %ld\n", pq_size);
        //printf("omega size: %ld\n", pq_size);
        //printf("u size: %ld\n", u_size);
        //printf("lgp: %d, deg: %d, max_deg: %d\n", p.lgp, p.deg, p.max_deg);
    }

    instantiate_constants();
    CUDA_CHECK(hipDeviceSynchronize());

    hipEventRecord(start, 0);

    uint log_p = 0;
    while (log_p < p.lgn) {
        uint deg = std::min(p.max_deg, p.lgn - log_p);
        uint32_t threadsPerBlock =
            1 << std::min(deg - 1, (uint32_t)MAX_LOG2_LOCAL_WORK_SIZE);
        uint32_t blocksPerGrid = (p.n >> deg);
        size_t u_size = sizeof(Fr) * (1 << deg);

        radix_fft<<<blocksPerGrid, threadsPerBlock, u_size>>>(
            d_x, d_y, d_pq, d_omegas, p.n, log_p, deg, p.max_deg);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipGetLastError());

        log_p += deg;
        Fr* tmp = d_x;
        d_x = d_y;
        d_y = tmp;
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed=0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("FFT Kernel time: %fms\n", elapsed);

    CUDA_CHECK(hipMemcpyAsync(p.x, d_x, x_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_pq));
    CUDA_CHECK(hipFree(d_omegas));

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipFree(d_x));
    hipCtxDestroy(context);

    return Compute_Ok;
}

// extern "C" cannot use templates.
extern "C" {
State G1_multiexp_cuda(G1InputParameters p) {
    return multiexp_cuda<G1>(p);
}

State G2_multiexp_cuda(G2InputParameters p) {
    return multiexp_cuda<G2>(p);
}

uint64_t G1_multiexp_chunk_size(G1InputParameters p) {
    return multiexp_chunk_size<G1>(p);
}

uint64_t G2_multiexp_chunk_size(G2InputParameters p) {
    return multiexp_chunk_size<G2>(p);
}

State Fr_radix_fft(FFTInputParameters p) {
     return radix_fft_cuda(p);
}

}
